#include "hip/hip_runtime.h"
// Includes.
#include <assert.h>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>


// Macros.
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

#define gpuErrchk(ans)                      \
{                                           \
    gpuAssert((ans), __FILE__, __LINE__);   \
}

extern "C"
{
  #include "utils.h"
}

// Dataset size.
#ifndef N
  #define N 10000
#endif

// Block size.
#ifndef BLOCK_SIZE
  #define BLOCK_SIZE 32
#endif

// Data type.
#ifndef DATA_T
  #define DATA_T double
#endif


// Utility per scambiare tra loro due puntatori (y_old/y_new).
void swapPointers(DATA_T *__restrict__(&ptr1), DATA_T *__restrict__(&ptr2)) {
  DATA_T *__restrict__ temp = ptr1;
  ptr1 = ptr2;
  ptr2 = temp;
}


// Host kernel.
void kernel_durbin_host(DATA_T * r, DATA_T * out)
{
  int i, k;
  DATA_T sum, alpha, beta;
  DATA_T y[2][N];
  alpha = r[0];
  beta = 1;
  y[0][0] = r[0];

  for (k = 1; k < N; k++)
  {
    beta = beta - alpha * alpha * beta;
    sum = r[k];

    for (i = 0; i <= k - 1; i++)
      sum += r[k - i - 1] * y[(k - 1) % 2][i];

    alpha = -sum * beta;

    for (i = 0; i <= k - 1; i++)
      y[k % 2][i] = y[(k - 1) % 2][i] + alpha * y[(k - 1) % 2][k - i - 1];
    
    y[k % 2][k] = alpha;
  }

  for (i = 0; i < N; i++)
    out[i] = y[(N - 1) % 2][i];
}


// Device kernel.
// Variabili dislocate su device.
__device__ DATA_T d_alpha, d_beta, d_sum;

// Primo kernel -> calcolo del nuovo beta + calcolo delle somme parziali e successiva reduction su sum.
__global__ void first_kernel(DATA_T *__restrict__ y, DATA_T *__restrict__ r, int k)
{
  __shared__ DATA_T partialSum[BLOCK_SIZE];

  // Coordinate del thread.
  int tid = threadIdx.x;
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  
  // Calcolo del valore base di sum.
  d_sum = r[k];

  // Calcolo del nuovo beta.
  DATA_T beta = d_beta;
  d_beta = beta - d_alpha * d_alpha * beta;

  // Caricamento delle somme parziali in memoria condivisa.
  if (i < k)
    partialSum[tid] = r[k - i - 1] * y[i];
  else
    partialSum[tid] = 0;

  __syncthreads();

  // Riduzione. Ciascun blocco porta la propria somma parziale in partialSum[0].
  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
  {
    if (tid < stride)
      partialSum[tid] += partialSum[tid + stride];
      
    __syncthreads();
  }

  // Il thread con tid 0 di ciascun blocco aggiorna il valore globale nel device con una atomicAdd.
  if (tid == 0)
    atomicAdd(&d_sum, partialSum[0]);
}

// Secondo kernel -> calcolo del nuovo alpha + calcolo del nuovo y in stile saxpy.
__global__ void second_kernel(DATA_T *__restrict__ y_old, DATA_T *__restrict__ y_new, int k)
{
  // Coordinate del thread.
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  // Calcolo del nuovo alpha.
  d_alpha = -d_sum * d_beta;

  if (i < k)
    y_new[i] = y_old[i] + d_alpha * y_old[k - i - 1];

  y_new[k] = d_alpha;
       
}

// Funzione chiamante dei kernel. Replica il kernel di Durbin.
void kernel_durbin_device(
  DATA_T *__restrict__ y_old,
  DATA_T *__restrict__ y_new,
  DATA_T *__restrict__ d_r)
{
  int k;
  // int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
  int GRID_SIZE;

  for (k = 1; k < N; k++)
  {
    GRID_SIZE = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Calcolo del nuovo beta e di sum.    
    first_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(y_old, d_r, k);

    // Calcolo del nuovo alpha e del nuovo y.
    second_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(y_old, y_new, k);

    // Scambio degli y.
    swapPointers(y_old, y_new);
  }
}


int main(int argc, char **argv)
{
  // Default data structures.
  int iret = 0;
  struct timespec rt[2];
  DATA_T wt;

  // Algorithm data structures.
  DATA_T *h_r, *h_out, *d_out;

  if (NULL == (h_r = (DATA_T *)malloc(sizeof(*h_r) * N)))
  {
    printf("error: memory allocation for 'h_r'\n");
    iret = -1;
  }
  if (NULL == (h_out = (DATA_T *)malloc(sizeof(*h_out) * N)))
  {
    printf("error: memory allocation for 'h_out'\n");
    iret = -1;
  }
  if (NULL == (d_out = (DATA_T *)malloc(sizeof(*d_out) * N)))
  {
    printf("error: memory allocation for 'd_out'\n");
    iret = -1;
  }

  // Device data structures.
  DATA_T *d_r, *y_old, *y_new;

  // Device mallocs.
  gpuErrchk(hipMalloc((void **)&d_r, sizeof(DATA_T) * N));
  gpuErrchk(hipMalloc((void **)&y_old, sizeof(DATA_T) * N));
  gpuErrchk(hipMalloc((void **)&y_new, sizeof(DATA_T) * N));

  // Return if any error occurred in mallocs.
  if (0 != iret)
  {
    free(h_r);
    free(h_out);
    free(d_out);
    gpuErrchk(hipFree(d_r));
    gpuErrchk(hipFree(y_old));
    gpuErrchk(hipFree(y_new));
    exit(EXIT_FAILURE);
  }

  // Init data.
  int i;

  #pragma omp parallel for
  for (i = 0; i < N; i++)
    h_r[i] = (DATA_T)(i + 1) / N / 4.0;

  // Test host.
  // Start timer.
  clock_gettime(CLOCK_REALTIME, rt + 0);

  // Function kernel durbin call (host).
  kernel_durbin_host(h_r, h_out);

  // Stop timer.
  clock_gettime(CLOCK_REALTIME, rt + 1);

  // Print results.
  wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
  printf("Durbin (Host) : %9.3f sec %9.1f GFLOPS\n", wt, 2.0 * N * N * N / (1.0e9 * wt));

  // Test device.
  // Start timer.
  clock_gettime(CLOCK_REALTIME, rt + 0);

  // Memcopies.
  // Device's array r.
  gpuErrchk(hipMemcpy(d_r, h_r, sizeof(DATA_T) * N, hipMemcpyHostToDevice));
  // y_old[0] = r[0].
  gpuErrchk(hipMemcpy(y_old, d_r, sizeof(DATA_T), hipMemcpyDeviceToDevice));
  DATA_T alpha;
  DATA_T beta = 1;
  // alpha = r[0].
  gpuErrchk(hipMemcpy(&alpha, d_r, sizeof(DATA_T), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_alpha), &alpha, sizeof(DATA_T)));
  // beta = 1.
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_beta), &beta, sizeof(DATA_T)));

  // Function kernel durbin call (device).
  kernel_durbin_device(y_old, y_new, d_r);

  // out = y_new dell'ultima iterazione di durbin (viene swappato -> quindi y_old).
  gpuErrchk(hipMemcpy(d_out, y_new, sizeof(DATA_T) * N, hipMemcpyDeviceToHost));

  // Debug.
  for (i = 0; i < N; i++)
    printf("out[%d] = %f\n", i, d_out[i]);

  // Stop timer.
  clock_gettime(CLOCK_REALTIME, rt + 1);

  // Print results.
  wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
  printf("Durbin (GPU): %9.3f sec %9.1f GFLOPS\n", wt, 2.0 * N * N * N / (1.0e9 * wt));

  // Frees.
  free(h_r);
  free(h_out);
  free(d_out);
  gpuErrchk(hipFree(d_r));
  gpuErrchk(hipFree(y_old));
  gpuErrchk(hipFree(y_new));

  return 0;
}
